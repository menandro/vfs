#include "hip/hip_runtime.h"
#include "stereotgv.h"

texture<float, hipTextureType2D, hipReadModeElementType> gray_img;
texture<float, hipTextureType2D, hipReadModeElementType> imgToFilter;

// Calculate anisotropic diffusion tensor
__global__ 
void TgvCalcTensorKernel(float* gray, float beta, float gamma, int size_grad,
	float* atensor, float* btensor, float* ctensor,
	int width, int height, int stride)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;

		float dx = 1.0f / (float)width;
		float dy = 1.0f / (float)height;

		float x = ((float)ix + 0.5f) * dx;
		float y = ((float)iy + 0.5f) * dy;

		float2 grad;
		float t0;
		// x derivative
		t0 = tex2D(gray_img, x + 1.0f * dx, y);
		t0 -= tex2D(gray_img, x, y);
		t0 = tex2D(gray_img, x + 1.0f * dx, y + 1.0f * dy);
		t0 -= tex2D(gray_img, x, y + 1.0f * dy);
		grad.x = t0;

		// y derivative
		t0 = tex2D(gray_img, x, y + 1.0f * dy);
		t0 -= tex2D(gray_img, x, y);
		t0 = tex2D(gray_img, x + 1.0f * dx, y + 1.0f * dy);
		t0 -= tex2D(gray_img, x + 1.0f * dx, y);
		grad.y = t0;

		float min_n_length = 1e-8f;
		float min_tensor_val = 1e-8f;

		float abs_img = sqrtf(grad.x*grad.x + grad.y*grad.y);
		float norm_n = abs_img;

		float2 n_normed;
		n_normed.x = grad.x / norm_n;
		n_normed.y = grad.y / norm_n;

		if (norm_n < min_n_length) {
			n_normed.x = 1.0f;
			n_normed.y = 0.0f;
		}

		float2 nT_normed;
		nT_normed.x = n_normed.y;
		nT_normed.y = -n_normed.x;

		float wtensor;
		if (expf(-beta * powf(abs_img, gamma)) > min_tensor_val) {
			wtensor = expf(-beta * powf(abs_img, gamma));
		}
		else wtensor = min_tensor_val;

		float a = wtensor * n_normed.x * n_normed.x + nT_normed.x * nT_normed.x;
		float c = wtensor * n_normed.x * n_normed.y + nT_normed.x * nT_normed.y;
		float b = wtensor * n_normed.y * n_normed.y + nT_normed.y * nT_normed.y;
		atensor[pos] = a;
		btensor[pos] = b;
		ctensor[pos] = c;
	}
}


void StereoTgv::CalcTensor(float* gray, float beta, float gamma, int size_grad,
	int w, int h, int s,
	float* a, float* b, float* c)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(h, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	gray_img.addressMode[0] = hipAddressModeMirror;
	gray_img.addressMode[1] = hipAddressModeMirror;
	gray_img.filterMode = hipFilterModeLinear;
	gray_img.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, gray_img, gray, w, h, s * sizeof(float));

	TgvCalcTensorKernel << < blocks, threads >> > (gray, beta, gamma, size_grad,
		a, b, c, w, h, s);
}


// Calculate anisotropic diffusion tensor
__global__ void TgvGaussianKernel(float* input, float* output,
	int width, int height, int stride)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;

		float dx = 1.0f / (float)width;
		float dy = 1.0f / (float)height;

		float x = ((float)ix + 0.5f) * dx;
		float y = ((float)iy + 0.5f) * dy;

		float2 grad;
		float t0 = (1 / 4.0f)*tex2D(imgToFilter, x, y);
		t0 += (1 / 16.0f)*tex2D(imgToFilter, x - 1.0f * dx, y - 1.0f * dy);
		t0 += (1 / 16.0f)*tex2D(imgToFilter, x - 1.0f * dx, y + 1.0f * dy);
		t0 += (1 / 16.0f)*tex2D(imgToFilter, x + 1.0f * dx, y - 1.0f * dy);
		t0 += (1 / 16.0f)*tex2D(imgToFilter, x + 1.0f * dx, y + 1.0f * dy);
		t0 += (1 / 8.0f)*tex2D(imgToFilter, x - 1.0f * dx, y);
		t0 += (1 / 8.0f)*tex2D(imgToFilter, x + 1.0f * dx, y);
		t0 += (1 / 8.0f)*tex2D(imgToFilter, x, y - 1.0f * dy);
		t0 += (1 / 8.0f)*tex2D(imgToFilter, x, y + 1.0f * dy);

		output[pos] = t0;
	}
}


void StereoTgv::Gaussian(float* input, int w, int h, int s, float* output)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	imgToFilter.addressMode[0] = hipAddressModeMirror;
	imgToFilter.addressMode[1] = hipAddressModeMirror;
	imgToFilter.filterMode = hipFilterModeLinear;
	imgToFilter.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, imgToFilter, input, w, h, s * sizeof(float));
	TgvGaussianKernel << < blocks, threads >> > (input, output,
		w, h, s);
}


// **************************
//  MASKED
// **************************
// Calculate anisotropic diffusion tensor
__global__
void TgvCalcTensorMaskedKernel(float* gray, float* mask, float beta, float gamma, int size_grad,
	float* atensor, float* btensor, float* ctensor,
	int width, int height, int stride)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	float2 grad;
	float t0;
	// x derivative
	t0 = tex2D(gray_img, x + 1.0f * dx, y);
	t0 -= tex2D(gray_img, x, y);
	t0 = tex2D(gray_img, x + 1.0f * dx, y + 1.0f * dy);
	t0 -= tex2D(gray_img, x, y + 1.0f * dy);
	grad.x = t0;

	// y derivative
	t0 = tex2D(gray_img, x, y + 1.0f * dy);
	t0 -= tex2D(gray_img, x, y);
	t0 = tex2D(gray_img, x + 1.0f * dx, y + 1.0f * dy);
	t0 -= tex2D(gray_img, x + 1.0f * dx, y);
	grad.y = t0;

	float min_n_length = 1e-8f;
	float min_tensor_val = 1e-8f;

	float abs_img = sqrtf(grad.x*grad.x + grad.y*grad.y);
	float norm_n = abs_img;

	float2 n_normed;
	n_normed.x = grad.x / norm_n;
	n_normed.y = grad.y / norm_n;

	if (norm_n < min_n_length) {
		n_normed.x = 1.0f;
		n_normed.y = 0.0f;
	}

	float2 nT_normed;
	nT_normed.x = n_normed.y;
	nT_normed.y = -n_normed.x;

	float wtensor;
	if (expf(-beta * powf(abs_img, gamma)) > min_tensor_val) {
		wtensor = expf(-beta * powf(abs_img, gamma));
	}
	else wtensor = min_tensor_val;

	float a = wtensor * n_normed.x * n_normed.x + nT_normed.x * nT_normed.x;
	float c = wtensor * n_normed.x * n_normed.y + nT_normed.x * nT_normed.y;
	float b = wtensor * n_normed.y * n_normed.y + nT_normed.y * nT_normed.y;
	atensor[pos] = a;
	btensor[pos] = b;
	ctensor[pos] = c;
}


void StereoTgv::CalcTensorMasked(float* gray, float* mask, float beta, float gamma, int size_grad,
	int w, int h, int s,
	float* a, float* b, float* c)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	gray_img.addressMode[0] = hipAddressModeMirror;
	gray_img.addressMode[1] = hipAddressModeMirror;
	gray_img.filterMode = hipFilterModeLinear;
	gray_img.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, gray_img, gray, w, h, s * sizeof(float));

	TgvCalcTensorMaskedKernel << < blocks, threads >> > (gray, mask, beta, gamma, size_grad,
		a, b, c, w, h, s);
}


// Calculate anisotropic diffusion tensor MASKED
__global__ void TgvGaussianMaskedKernel(float* input, float* mask, float* output,
	int width, int height, int stride)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	float2 grad;
	float t0 = (1 / 4.0f)*tex2D(imgToFilter, x, y);
	t0 += (1 / 16.0f)*tex2D(imgToFilter, x - 1.0f * dx, y - 1.0f * dy);
	t0 += (1 / 16.0f)*tex2D(imgToFilter, x - 1.0f * dx, y + 1.0f * dy);
	t0 += (1 / 16.0f)*tex2D(imgToFilter, x + 1.0f * dx, y - 1.0f * dy);
	t0 += (1 / 16.0f)*tex2D(imgToFilter, x + 1.0f * dx, y + 1.0f * dy);
	t0 += (1 / 8.0f)*tex2D(imgToFilter, x - 1.0f * dx, y);
	t0 += (1 / 8.0f)*tex2D(imgToFilter, x + 1.0f * dx, y);
	t0 += (1 / 8.0f)*tex2D(imgToFilter, x, y - 1.0f * dy);
	t0 += (1 / 8.0f)*tex2D(imgToFilter, x, y + 1.0f * dy);

	output[pos] = t0;
}


void StereoTgv::GaussianMasked(float* input, float* mask, int w, int h, int s, float* output)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	imgToFilter.addressMode[0] = hipAddressModeMirror;
	imgToFilter.addressMode[1] = hipAddressModeMirror;
	imgToFilter.filterMode = hipFilterModeLinear;
	imgToFilter.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, imgToFilter, input, w, h, s * sizeof(float));
	TgvGaussianMaskedKernel << < blocks, threads >> > (input, mask, output,
		w, h, s);
}